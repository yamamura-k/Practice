
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void add_vector(float *out, float *a, float *b, int n)
{
    for( int i = 0 ; i < n ; ++i )
    {
        out[i] = a[i] + b[i];
    }
}

int main()
{
    const int N = 10;
    float *a, *b, *out;
    float *cuda_a, *cuda_b, *cuda_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    for( int i = 0 ; i < N ; i++ )
    {
        a[i] = i*1.1;
        b[i] = N*0.6-i;
        out[i] = 0.0;
    }
    hipMalloc((void**)&cuda_a, sizeof(float) * N);
    hipMalloc((void**)&cuda_b, sizeof(float) * N);
    hipMalloc((void**)&cuda_out, sizeof(float) * N);
    hipMemcpy(cuda_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    add_vector<<<1,1>>>(cuda_out, cuda_a, cuda_b, N);
    for( int i = 0 ; i < N ; ++i )
    {
      std::cout << out[i] << "  ";
    }
    std::cout << std::endl;
    hipFree(cuda_a);
    free(a);
    return 0;
}
