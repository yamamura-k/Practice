#include <iostream>
#include <hip/hip_runtime.h>


__global__ void add_vector(float *out, float *a, float *b, int n)
{
    int index = threadIdx.x;
    int stride = blockDim.x;

    for( int i = index ; i < n ; i += stride )
    {
        out[i] = a[i] + b[i];
    }
}

int main()
{
    const int N = 100000;
    float *a, *b, *out;
    float *cuda_a, *cuda_b, *cuda_out;

    std::cout << "This program computes sum of " << N << " dimensional vectors." << std::endl;
    std::cout << "All variables are written in source code." << std::endl;

    // allocate memory on cpu for vector a, b, out
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    
    // initialization
    for( int i = 0 ; i < N ; i++ )
    {
        a[i] = i*1.1;
        b[i] = N*0.6-i;
        out[i] = 0.0;
    }

    // allocate memory on gpu for vector cuda_a, cuda_b, cuda_out
    hipMalloc((void**)&cuda_a, sizeof(float) * N);
    hipMalloc((void**)&cuda_b, sizeof(float) * N);
    hipMalloc((void**)&cuda_out, sizeof(float) * N);

    // copy memory from cpu to gpu
    hipMemcpy(cuda_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    // main procedure
    // In this case, a kernel launches with a grid of 1 thread blocks 
    // and each thread block has 256 parallel threads.
    add_vector<<<1, 256>>>(cuda_out, cuda_a, cuda_b, N);

    // copy result from gpu to cpu
    hipMemcpy(out, cuda_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    //Verification
    std::cout << "result: ";
    for( int i = 0 ; i < N ; ++i )
    {
      std::cout << out[i] << "  ";
    }
    std::cout << std::endl;
    std::cout << "vector1: ";
    for( int i = 0 ; i < N ; ++i )
    {
      std::cout << a[i] << "  ";
    }
    std::cout << std::endl;
    std::cout << "vector2: ";
    for( int i = 0 ; i < N ; ++i )
    {
      std::cout << b[i] << "  ";
    }
    std::cout << std::endl;
    // release unused memories
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_out);
    free(a);
    free(b);
    free(out);
    return 0;
}
